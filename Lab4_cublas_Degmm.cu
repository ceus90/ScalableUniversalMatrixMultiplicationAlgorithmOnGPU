#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <unistd.h>
#include <errno.h>
#include <sys/types.h>
#include <memory.h>
#include <malloc.h>
#include <math.h>
#include <stddef.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#include <hipblas.h>

extern "C" void launch_multiply(int bsize, int n, double *A, double *B, double *C)
{
double *Ad, *Bd, *Cd;

//Allocating device memory on the GPU for the matrices
hipMalloc(&Ad,(size_t)(bsize*bsize*sizeof(double)));
hipMalloc(&Bd,(size_t)(bsize*bsize*sizeof(double)));
hipMalloc(&Cd,(size_t)(bsize*bsize*sizeof(double)));

//Copy A and B from host memory to device memory
hipMemcpy(Ad,A,bsize*bsize*sizeof(double),hipMemcpyHostToDevice);
hipMemcpy(Bd,B,bsize*bsize*sizeof(double),hipMemcpyHostToDevice);

int lda = n, ldb = n, ldc = n;
const double alf = 1;
const double bet = 0;
const double *alpha = &alf;
const double *beta = &bet;
hipblasHandle_t handle;
hipblasCreate(&handle);
hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, alpha, Ad, lda, Bd, ldb, beta, Cd, ldc);
hipblasDestroy(handle);

hipFree(Ad);
hipFree(Bd);
hipFree(Cd);
}
